#include "hip/hip_runtime.h"

#define GPU __HIPCC__

#include <string>
#ifdef GPU
#	include <hip/hip_runtime.h>
#else
#	include <omp.h>
#	include <math.h>
#	include <ctime>
#endif

//
// parameters
//
#ifdef GPU
#	define nThreads   64
#	define nBlocks    84
#	define nLoop      16
#else
#	define nThreads    1
#	define nBlocks     1
#	define nLoop      16
#endif
#define TT_SIZE   4194304
#define UCTK      0.44f
#define FPU       1.10f

//
// printf
//
#ifdef GPU
#	include "cuPrintf.cu"
#	define print(format, ...) cuPrintf(format, __VA_ARGS__)
#else
#	define print(format, ...) printf(format, __VA_ARGS__)
#endif

//
// locks
//
#ifdef GPU
#	define LOCK          int
#	define l_create(x)   ((x) = 0)
#	define l_trylock(x)  (atomicExch(&(x),1))
#	define l_lock(x)     while(l_trylock(x) != 0);
#	define l_unlock(x)   (atomicExch(&(x),0))
#	define l_add(x,v)	 (atomicAdd(&x,v))
#	define l_sub(x,v)	 (atomicSub(&x,v))
#	define l_barrier()   __syncthreads()
#else
#	define LOCK          omp_lock_t
#	define l_create(x)   omp_init_lock(&x)
#	define l_trylock(x)  omp_test_lock(&x)
#	define l_lock(x)     omp_set_lock(&x)
#	define l_unlock(x)   omp_unset_lock(&x)   
template <class T>
inline void l_add(T x,T v) { 
	#pragma omp atomic 
		x+=v;
}
template <class T>
inline void l_sub(T x,T v) { 
	#pragma omp atomic 
		x-=v;
}
inline void l_barrier() { 
	#pragma omp barrier 
}
#endif

//
// undef cuda specific code
//
#ifndef GPU
#	undef  __host__
#	undef  __device__
#   undef  __global__
#   undef  __shared__
#   undef  __constant__
#	define __host__
#	define __device__
#	define __global__
#   define __shared__
#   define __constant__
#if defined (__GNUC__)
#	define __align__(x)  __attribute__ ((aligned(x)))
#else
#	define __align__(x) __declspec(align(x))
#endif
#endif

//
// types
//
#ifdef _MSC_VER
	typedef unsigned __int64 U64;
	typedef unsigned int U32;
#	define U64(x) (x##ui64)
#	define FMTU64 "0x%016I64x"
#else
#   include <inttypes.h>
	typedef uint64_t U64;
	typedef uint32_t U32;
#	define U64(x) (x##ull)
#	define FMTU64 "0x%016llx"
#endif

//
// define board game
//
typedef U64 MOVE;

struct BOARD {
	U64 wpawns;
	U64 all;
	U32 randn;
	char player;
	char emptyc;

	U32 playout(const BOARD&);
	void make_random_move();
	bool is_white_win();

	__device__ __host__
	void clear() {
		wpawns = 0;
		all = U64(0xffffffffffffffff);
		emptyc = 64;
		player = 0;
	}

	__host__ __device__
	void copy(const BOARD& b) {
		wpawns = b.wpawns;
		all = b.all;
		player = b.player;
		emptyc = b.emptyc;
	}

	__device__ __host__
	void do_move(const MOVE& move) {
		all ^= move;
		if(player == 0)
			wpawns ^= move;
		player ^= 1;
		emptyc--;
	}

	__device__ __host__
	void seed(int sd) {
		randn = sd;
	}

	__device__ __host__
	U32 rand() {
		randn *= 214013;
		randn += 2531011;
		return ((randn >> 16) & 0x7fff);
	}

	__device__ __host__
	U64 rand64() {
		return((U64)rand()) ^ 
			  ((U64)rand() << 15) ^ ((U64)rand() << 30) ^
			  ((U64)rand() << 45) ^ ((U64)rand() << 60);
	}
};


__device__ __host__
void BOARD::make_random_move() {
	U32 rbit = rand() % emptyc;
	U64 mbit = all;
	for(U32 i = 0;i < rbit;i++)
		mbit &= mbit - 1; 
	mbit = mbit & -mbit;

	if(player == 0)
		wpawns ^= mbit;
	all ^= mbit;
	player ^= 1;
	emptyc--;
}

__device__ __host__
bool BOARD::is_white_win(){
	U64 m = (wpawns & U64(0x00000000000000ff)),oldm;
	do {
		oldm = m;
		m |=((((m << 8) | (m >> 8)) | 
			 (((m << 9) | (m << 1)) & U64(0xfefefefefefefefe)) | 
			 (((m >> 9) | (m >> 1)) & U64(0x7f7f7f7f7f7f7f7f))) 
			 & wpawns
			);
		if(m & U64(0xff00000000000000)) {
			return true;
		}
	} while(m != oldm);
	return false;
}

__device__ __host__
U32 BOARD::playout(const BOARD& b) {
	U32 wins = 0;
	for(U32 i = 0;i < nLoop;i++) {
		this->copy(b);

		while(emptyc > 0)
			make_random_move();
			
		if(is_white_win())
			wins++;
	}
	return wins;
}

__constant__
unsigned int index64[64];

__device__ __host__
unsigned int firstone(U64 bb) {
	unsigned int folded;
	bb ^= bb - 1;
	folded = (int) bb ^ (bb >> 32);
	return index64[folded * 0x78291ACF >> 26];
}

//
//sq to string and vice versa
//
#define file(x)          ((x) & 7)
#define rank(x)          ((x) >> 3)
#define SQ(x,y)          (((x) << 3) + (y))

__device__ __host__
char* sq_str(const int& sq,char* s) {
	int f = file(sq);
	int r = rank(sq);
	*s++ = 'a' + (f);
	*s++ = '1' + (r);
	*s = 0;
	return s;
}

__host__
const char* str_sq(int& sq,const char* is) {
	const char* s = is;
	int f = tolower(*s++) - 'a';
	int r = atoi(s++) - 1;
	sq = SQ(r,f);
	return s;
}

//
// Node
//

struct Node {
	MOVE move;
	U32 uct_wins;
	U32 uct_visits;
	Node* parent;
	Node* child;
	Node* next;
	LOCK lock;
	int workers;
	
	__device__ __host__
	void clear() {
		uct_wins = 0;
		uct_visits = 0;
		parent = 0;
		child = 0;
		next = 0;
		move = MOVE();
		l_create(lock);
		workers = 0;
	}
};

//
// Table
//

namespace TABLE {
	__device__ Node* mem_;
	__device__ int tsize;
	__device__ BOARD root_board;
	__device__ Node* root_node;
	__device__ Node* head;
	__device__ int size;
	__device__ LOCK lock;
	Node* hmem_;

	__device__ Node* get_node() {
		if(size > 0) {	
			l_lock(lock);
			if(size > 0) {
				size--;
				head++;
				head->clear();
			} else 
				head = 0;
			l_unlock(lock);
			return head;
		} else {
			return 0;
		}
	}
	__global__ void reset() {
		head = mem_;
		size = tsize;
		root_node = get_node();
	}
	
	__global__ void print_tree(int depthLimit) {
		int depth = 0,max_depth = 0,average_depth = 0;
		int leaf_nodes = 0,total_nodes = 0;
		char str[4];
		int sq;
		Node* current = root_node;
		while(current) {
			while(current) {
				while(current) {

					if(current->uct_visits && depth <= depthLimit) {
						for(int i = 0;i < depth;i++)
							print("\t");
						sq = firstone(current->move);
						sq_str(sq,str);
						print("%d.%s %d %d %.6f\n",
							depth,(const char*)str,
							current->uct_wins,current->uct_visits,
							float(current->uct_wins) / current->uct_visits
							);
					}

					total_nodes++;
					if(current->child) {
						depth++;
						current = current->child;
					} else {
						if(depth > max_depth)
							max_depth = depth;
						average_depth += depth;
						leaf_nodes++;
						break;
					}
				}
NEXT:
				if(current->next) {
					current = current->next;
				} else break;
			}
			if(current->parent) {
				depth--;
				current = current->parent;
				goto NEXT;
			} else {
				break;
			}
		}

		print("Total nodes   : %d\n",total_nodes);
		print("Leaf  nodes   : %d\n",leaf_nodes);
		print("Maximum depth : %d\n",max_depth);
		print("Average depth : %.2f\n",average_depth / float(leaf_nodes));
	}
	__device__ void create_children(BOARD* b,Node* n) {
		l_lock(n->lock);
		if(n->child) {
			l_unlock(n->lock);
			return;
		}

		Node* last = n;
		U64 m = b->all;
		U64 lsb;
		while(m) {
			lsb = m & -m;
		
			Node* node = get_node();
			if(!node) break;
			node->move = lsb;
			node->parent = n;
			if(last == n) last->child = node;
			else last->next = node;
			last = node;
			
			m ^= lsb;
		}

		l_unlock(n->lock);
	}

	__device__ Node* UCT_select(Node* n) {
		Node* bnode = 0;
		Node* current = n->child;
		float bvalue = -1.f,value;
		float logn = logf(float(n->uct_visits + 1));
		while(current) {
			if(current->uct_visits > 0) { 
				value = UCTK * sqrtf(logn / (current->uct_visits + 1))
					+ (current->uct_wins + 1) / (current->uct_visits + 1); 
			} else {
				value = FPU;
			}

			value -= (current->workers / 128.f);

			if(value > bvalue) {
				bvalue = value;
				bnode = current;
			}
			current = current->next;
		}
		return bnode;
	}

	__host__ void allocate(int N) {
		static const unsigned int mindex64[64] = {
			63, 30,  3, 32, 59, 14, 11, 33,
			60, 24, 50,  9, 55, 19, 21, 34,
			61, 29,  2, 53, 51, 23, 41, 18,
			56, 28,  1, 43, 46, 27,  0, 35,
			62, 31, 58,  4,  5, 49, 54,  6,
			15, 52, 12, 40,  7, 42, 45, 16,
			25, 57, 48, 13, 10, 39,  8, 44,
			20, 47, 38, 22, 17, 37, 36, 26
		};
#ifdef GPU
		hipMalloc((void**) &hmem_,N * sizeof(Node));
		hipMemcpyToSymbol(HIP_SYMBOL(tsize),&N,sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(mem_),&hmem_,sizeof(Node*));
		hipMemcpyToSymbol(HIP_SYMBOL(index64),mindex64,sizeof(mindex64));
#else
		hmem_ = (Node*) malloc(N * sizeof(Node));
		tsize = N;
		mem_ = hmem_;
		memcpy(index64,mindex64,sizeof(mindex64));
		l_create(lock);
#endif
	}
	__host__ void release() {
#ifdef GPU
		hipFree(hmem_);
#else
		free(hmem_);
#endif
	}
}

//
// playout
//
__global__ 
void playout(U32 N) {
	//
	// create blocks
	//
#ifdef GPU
	{
		const int blockId = blockIdx.x;
#else
#pragma omp parallel num_threads(nBlocks)
	{
		const int blockId = omp_get_thread_num();
#endif
		//
		//shared data with in a block
		//
		__shared__ U32 cache[nThreads];
		__shared__ BOARD sb;
		__shared__ Node* n;
		__shared__ bool finished;

		//
		// create threads and allocate a BOARD on register
		//
#ifdef GPU
		{
			const int threadId = threadIdx.x;
#else
#pragma omp parallel num_threads(nThreads)
		{
			const int threadId = omp_get_thread_num();
			print("Block %d : Thread %d of %d\n",
				blockId,threadId,nThreads);
#endif
			BOARD b;
			b.seed(blockId * nBlocks + threadId);
			//
			//loop forever
			//
			while(true) {

				//get node
				if(threadId == 0) {
					finished = false;
					n = TABLE::root_node;
					sb.copy(TABLE::root_board);

					while(n->child) {
						n = TABLE::UCT_select(n);
						sb.do_move(n->move);
					}

					if(n->uct_visits) {
						TABLE::create_children(&sb,n);
						Node* next = TABLE::UCT_select(n);
						if(next) {
							sb.do_move(next->move);
							n = next;
						}
					}

					l_add(n->workers,1);
				}
				l_barrier();
				b.copy(sb);

				//playout the position
				cache[threadId] = b.playout(sb);

				//reduction 
				l_barrier();
				int p = nThreads;
				int i = (p + 1) / 2;
				while (i != 0) {
					if ((threadId < i) && (threadId + i < p))
						cache[threadId] += cache[threadId + i];
					l_barrier();
					p = i;
					if(p > 1) i = (p + 1) / 2;
					else i = p / 2;
				}

				//update result
				if (threadId == 0) {
					l_sub(n->workers,1);

					U32 score;
					if(sb.player != 0) 
						score = cache[0];
					else
						score = nLoop * nThreads - cache[0];
						
					Node* current = n;
					while(current) {
						l_lock(current->lock);
						current->uct_wins += score;
						current->uct_visits += nLoop * nThreads;
						l_unlock(current->lock);
						score = nLoop * nThreads - score;
						current = current->parent;
					}
					if(TABLE::root_node->uct_visits >= N)
						finished = true;
				}
				l_barrier();
				if(finished)
					break;
			}
			//
			// end of work
			//
		}
	}
}
//
// GPU specific code
//

#ifdef GPU

__host__ 
void simulate(BOARD* b,U32 N) {
	hipMemcpyToSymbol(HIP_SYMBOL(TABLE::root_board),b,
		sizeof(BOARD),0,hipMemcpyHostToDevice);

	TABLE::reset <<<1,1>>> ();
	playout <<<nBlocks,nThreads>>> (N); 
	TABLE::print_tree <<<1,1>>> (1);

	cudaPrintfDisplay();
	printf("Errors: %s\n", 
		hipGetErrorString(hipPeekAtLastError()));
}
__host__
void init_device() {
	int count;
	hipDeviceProp_t prop;
	hipGetDeviceCount( &count );
	for (int i=0; i< count; i++) {
		hipGetDeviceProperties( &prop, i );
		printf( " --- General Information for device %d ---\n", i );
		printf( "Name: %s\n", prop.name );
		printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
		printf( "Clock rate: %d\n", prop.clockRate );
		printf( "Device copy overlap: " );
		if (prop.deviceOverlap)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );
		printf( "Kernel execition timeout : " );
		if (prop.kernelExecTimeoutEnabled)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );
		printf( " --- Memory Information for device %d ---\n", i );
		printf( "Total global mem: %ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem: %ld\n", prop.totalConstMem );
		printf( "Max mem pitch: %ld\n", prop.memPitch );
		printf( "Texture Alignment: %ld\n", prop.textureAlignment );
		printf( " --- MP Information for device %d ---\n", i );
		printf( "Multiprocessor count: %d\n",
			prop.multiProcessorCount );
		printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
		printf( "Registers per mp: %d\n", prop.regsPerBlock );
		printf( "Threads in warp: %d\n", prop.warpSize );
		printf( "Max threads per block: %d\n",
			prop.maxThreadsPerBlock );
		printf( "Max thread dimensions: (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1],
			prop.maxThreadsDim[2] );
		printf( "Max grid dimensions: (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1],
			prop.maxGridSize[2] );
		printf( "\n" );
	}

	printf("nBlocks=%d X nThreads=%d\n",nBlocks,nThreads);
	cudaPrintfInit();
	TABLE::allocate(TT_SIZE);
}
__host__ 
void finalize_device() {
	cudaPrintfEnd();
	TABLE::release();
}

#else

//
// Cpu specific code
//

__host__
void simulate(BOARD* bo,U32 N) {
	TABLE::root_board = *bo;
	TABLE::reset();
	playout(N);
	TABLE::print_tree(1);
}
__host__
void init_device() {
	omp_set_nested(1);
	omp_set_dynamic(0);
	TABLE::allocate(TT_SIZE);
}
__host__
void finalize_device() {
	TABLE::release();
}

#endif

//
// Test
//

__host__
void print_bitboard(U64 b) {
	std::string s = "";
	for(int i=7;i>=0;i--) {
		for(int z = 0; z < 7-i;z++)
			s += " ";
		for(int j=0;j<8;j++) {
			U64 m = (((U64)1) << (i * 8 + j));
			if(b & m) s += "1 ";
			else s += "0 ";
		}
		s += "\n";
	}
	printf("%s",s.c_str());
	printf("\n"FMTU64"\n\n",b);
}

static const char *const commands_recognized[] = {
	"d",
	"go",
	"quit",
	"help",
	NULL
};

int main() {
	init_device();

	BOARD b;
	b.clear();

	char str[64];
	while(true) {
		printf("$: ");
		scanf("%s",&str);
		if(!strcmp(str,"d")) {
			print_bitboard(b.wpawns);
			print_bitboard(b.all);
		} else if(!strcmp(str,"help")) {
			size_t index = 0;
			while (commands_recognized[index]) {
				puts(commands_recognized[index]);
				index++;
			}
		} else if(!strcmp(str,"go")) {
			clock_t start,end;
			start = clock();
			simulate(&b,128 * 28 * 128 * 100);
			end = clock();
			printf("time %d\n",end - start);
		} else if(!strcmp(str,"quit")) {
			break;
		} else {
			int move;
			str_sq(move,str);
			b.do_move((U64(1) << move));
		}
	}

	finalize_device();
}

//
// end
//

